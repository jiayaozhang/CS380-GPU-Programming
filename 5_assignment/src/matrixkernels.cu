#include "hip/hip_runtime.h"
#include <iostream>
#include "matrixkernels.cuh"
#include "hip/hip_runtime_api.h"


// our workstation's limit
#define BLOCK_XDIM 128
extern "C"
void Vprint(float* vec,int N){
	for (int i=0; i<N; i++){ 
		printf("%f ", vec[i]); 
	}	
	printf("\n");
}
extern "C"
void Vprintrange(float* vec,int S, int E){
	for (int i=S; i<E; i++){ 
		//printf(" i:%d val:%f ", i, vec[i]); 
		printf("%f ", vec[i]); 
	}	
	printf("\n");
}
extern "C"
int iDivUp( int a, int b ){
    return (a % b != 0) ? (a / b + 1) : (a / b);
}


extern "C"
unsigned int nextPow2( unsigned int x ) {
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}


// vector operation: x = fac0*a op fac1*b
__global__ void
_cl_vector_op_( int op, float fac0, float fac1, float *a, float *b, float *x, int dim )
{
	/* TASK 1.1: implement the elementwise vector operations
	
	 	x = fac0 * a (op) fac1 * b
	
		with op = {+,-,*, NONE}.
		NONE means x = fac0 * a
		
		HINT: remember to safeguard the index (the thread id might be larger than the array size)! 
		-> if the thread index is >= dim return!
		
	*/
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= dim) return;
	switch(op){
		case(-1): // NONE 
			x[idx] = a[idx] * fac0;
			break;
		case(0):  // ADD 
			x[idx] = a[idx] * fac0 + b[idx] * fac1;
			break;
		case(1):  // SUB 
			x[idx] = a[idx] * fac0 - b[idx] * fac1;
			break;
		case(2):  // MULT
			x[idx] = a[idx] * fac0 * b[idx] * fac1;
			break;
	}
}




// matrix vector multiplication: x = A*b op c
__global__ void
_cl_matrix_vector_( int op, float *A, float *b, float *c, float *x, int dim )
{
	/* TASK 1.2: implement the matrix vector multiplication
	
		x = A * b (op) c
	
		with op = {+,-,*,NONE}.
		NONE means x = A * b

		HINT: remember to safeguard the index (the thread id might be larger than the array size)!
		-> if the thread index is >= dim return!
	*/
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// load data in 
	float out = 0.0f;  /// dim = width * height   dim/ blockDim.x +1
	extern __shared__ char char_b[];
	float* s_b = (float*)char_b;
	for(int m=0; m<dim/blockDim.x+1; m++){
		int tmpidx = m*blockDim.x+threadIdx.x;
		if( tmpidx < dim) s_b[tmpidx] = b[tmpidx];
	}
	__syncthreads();
	if (idx >= dim) return; // should load the shared data and return !!!
	for(int i=0; i<dim; i++){
		out += A[idx * dim + i] * s_b[i];
	}
	switch(op){
		case(-1):
			x[idx] = out;
			break;
		case(0):
			x[idx] = out + c[idx];
			break;
		case(1):
			x[idx] = out - c[idx];
			break;
		case(2):
			x[idx] = out * c[idx];
			break;
	}
}


__global__ 
void _cl_vector_reduce_(int op, float* d_a, int dim){

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	for(unsigned int s=1; s<dim; s*=2){
		if (idx % (2*s) == 0 && idx + s < dim){  // step 1 , s= 1 ;
			
			//step 1 idx % 2 
			//step 2 idx %4
			//step 3 idx % 8
			//step etc..
			switch(op){
				case(0):  
					d_a[idx] += d_a[idx + s];  //sum 
					break;
				case(2):
					d_a[idx] *= d_a[idx + s];  // sum 
			}
		}
	}
}



// d_x = SUM[d_a * d_b]
float gpuReduceSUM( float* d_a, float *d_b, float* d_x, int dim, int nBlocks, int nThreads ){

	/* TASK 1.3: implement the vector multiplication and sum reduction

		d_x = SUM[d_a * d_b]
		
		implement reduction as discussed in the lecture using shared memory.
		
	*/
	float sum = 0.0f;

	_cl_vector_op_<<<nBlocks, nThreads>>>(CL_MULT, 1.0f, 1.0f, d_a, d_b, d_x, dim);
	checkCudaErrors( hipMemcpy( &sum, d_x, 1 * sizeof( float ), hipMemcpyDeviceToHost ) );
	printf("sum is %f \n", sum);

	checkCudaErrors( hipDeviceSynchronize() );

	_cl_vector_reduce_<<<nBlocks, nThreads>>>(CL_ADD, d_x, dim);

	checkCudaErrors( hipDeviceSynchronize() );
	
	checkCudaErrors( hipMemcpy( &sum, d_x, 1 * sizeof( float ), hipMemcpyDeviceToHost ) );
	printf("sum is %f \n", sum);
	return sum;
}

// x = A*a
extern "C" 
void multiplyMatrixVector( float *h_A, float *h_a, float *h_x, int dim )
{
	float *d_A, *d_a, *d_x;
	
	checkCudaErrors( hipMalloc( (void**) &d_A, dim * dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_a, dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_x, dim * sizeof( float ) ) );

	checkCudaErrors( hipMemcpy( d_A, h_A, dim * dim * sizeof( float ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy( d_a, h_a, dim * sizeof( float ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipMemcpy( d_x, h_x, dim * sizeof( float ), hipMemcpyHostToDevice ) );

	checkCudaErrors( hipDeviceSynchronize() );

	// x = A*a
	int nThreads = 128;
	int nBlocks = iDivUp( dim, nThreads );
	_cl_matrix_vector_<<< nBlocks, nThreads, dim*sizeof(float) >>>( NONE, d_A, d_a, NULL, d_x, dim );
	checkCudaErrors( hipDeviceSynchronize() );

	// copy solution from device to host
	checkCudaErrors( hipMemcpy( h_x, d_x, dim * sizeof( float ), hipMemcpyDeviceToHost ) );

	// release device memory
	checkCudaErrors( hipFree( d_A ) );
	checkCudaErrors( hipFree( d_a ) );
	checkCudaErrors( hipFree( d_x ) );
	
	
}

extern "C"
void checkright(float*a, float*b, int dim){
	for(int i=0; i<dim; i++){
		if(abs(a[i]-b[i]) > 0.01) printf("err check");
	}
}

extern "C"
void MMhost(float*A, float* x, float* out, int dim){
	for(int i=0; i<dim;i++) out[i] = 0.0f;
	for(int i=0; i<dim; i++)
		for(int j=0; j<dim; j++)
			out[i] += A[i*dim + j] * x[j];
}


extern "C" 
void computeConjugateGradient( float *h_A, float *h_b, float *h_x, int dim, float errorTolerance )
{
	int nThreads = 128;							// set the number of threads per block to use by default
	int nBlocks = iDivUp( dim, nThreads );

	float *d_A, *d_b, *d_x, *d_r, *d_p, *d_q, *d_tmp;
	float alpha, beta, rho = 0;
	//Vprintrange(h_A, 64*0, 64*1);

	//for(int i=0; i<dim; i++) h_x[i] = 1.0f;
	//Vprint(h_x, dim);
	//allocate device memory
	checkCudaErrors( hipMalloc( (void**) &d_A, dim * dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_b, dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_x, dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_r, dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_p, dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_q, dim * sizeof( float ) ) );
	checkCudaErrors( hipMalloc( (void**) &d_tmp, dim * sizeof( float ) ) );
	
	// copy host to device
	checkCudaErrors( hipMemcpy( d_A, h_A, dim * dim * sizeof( float ), hipMemcpyHostToDevice ) );

	checkCudaErrors( hipMemcpy( d_b, h_b, dim * sizeof(float), hipMemcpyHostToDevice ) );

	checkCudaErrors( hipMemcpy( d_x, h_x, dim * sizeof( float ), hipMemcpyHostToDevice ) );
	checkCudaErrors( hipDeviceSynchronize() );

	// init CG
	// ALGORITHM: r_0 = b-Ax_0
	// r_0 = Ax_0 - b
	
	_cl_matrix_vector_<<<nBlocks, nThreads, dim*sizeof(float)>>>( CL_SUB, d_A, d_x, d_b, d_r, dim );
	checkCudaErrors( hipDeviceSynchronize() );

//	hipError_t err = hipGetLastError();
//	 if (err != hipSuccess) 
//		 printf("Error: %s\n", hipGetErrorString(err));
//	float* h_r = (float*)malloc(dim*sizeof(float));
//	MMhost(h_A, h_x, h_r, dim);
//	checkCudaErrors( hipMemcpy( h_b, d_r, dim * sizeof( float ), hipMemcpyDeviceToHost ) );
//	printf("72th line\n");
//	Vprintrange(h_A, 200*72,200*73);

//	Vprintrange(h_b, 0, 128);
//	Vprintrange(h_r, 0, 128);
//	Vprintrange(h_b, 128, 200);
//	Vprintrange(h_r, 128, 200);
//	printf("check end\n");

	// r_0 = -r_0
	_cl_vector_op_<<< nBlocks, nThreads >>>( NONE, -1.0f, 0.0f, d_r, NULL, d_r, dim );


	// p_0 = r_0
	_cl_vector_op_<<< nBlocks, nThreads >>>( NONE,  1.0f, 0.0f, d_r, NULL, d_p, dim );
	checkCudaErrors( hipDeviceSynchronize() );

	// CG needs max dim iterations
	int i = 0;
	float minRho = 1000000000;
	for( i = 0; i < dim; i++ ){	
		
		// rho_k = sum(r_k * r_k)
		rho = gpuReduceSUM(d_r, d_r, d_tmp, dim, nBlocks, nThreads );
		checkCudaErrors( hipDeviceSynchronize() );
		
		if (minRho > rho) {
			minRho = rho;
		}
		
		//printf("iteration #%d, with rho = %f", i, rho);
		std::cout << "iteration #" << i << ", with rho = " << rho << "          " << '\r' << std::flush;
		// check here for criterion
		if( rho < errorTolerance) {
			break;
		}
		printf("rho %f \n", rho);	
		// q_k = A*p_k
		_cl_matrix_vector_<<< nBlocks, nThreads, dim*sizeof(float) >>>( NONE, d_A, d_p, NULL, d_q, dim );
		checkCudaErrors( hipDeviceSynchronize() );
		
		// alpha_k = rho_k / sum(p_k * q_k)
		alpha = rho / gpuReduceSUM(d_p, d_q, d_tmp, dim, nBlocks, nThreads );
		checkCudaErrors( hipDeviceSynchronize() );
		
		 // x_(k+1) = x_k + alpha_k * p_k
		_cl_vector_op_<<< nBlocks, nThreads >>>( CL_ADD, 1.0f, alpha, d_x, d_p, d_x, dim );
		checkCudaErrors( hipDeviceSynchronize() );
		
		// r_(k+1) = r_k + (-alpha_k * q_k)
		_cl_vector_op_<<< nBlocks, nThreads >>>( CL_ADD, 1.0f, -alpha, d_r, d_q, d_r, dim );
		checkCudaErrors( hipDeviceSynchronize() );

		// beta_k = sum(r_(k+1) * r_(k+1)) / rho_k
		beta = gpuReduceSUM(d_r, d_r, d_tmp, dim, nBlocks, nThreads ) / rho;
		checkCudaErrors( hipDeviceSynchronize() );
		
		// p_(k+1) = r_(k+1) + beta_k * p_k
		_cl_vector_op_<<< nBlocks, nThreads >>>( CL_ADD, 1.0f, beta, d_r, d_p, d_p, dim );
		checkCudaErrors( hipDeviceSynchronize() );
	}

	rho = gpuReduceSUM(d_r, d_r, d_tmp, dim, nBlocks, nThreads );

	printf("\nSolution found at iteration #%d, with rho = %f\n", i, rho);
	printf("\nminrho was %f\n", minRho);
	
	// copy solution from device to host
	checkCudaErrors( hipMemcpy( h_x, d_x, dim * sizeof( float ), hipMemcpyDeviceToHost ) );

	// release device memory
	checkCudaErrors( hipFree( d_A ) );
	checkCudaErrors( hipFree( d_b ) );
	checkCudaErrors( hipFree( d_x ) );
	checkCudaErrors( hipFree( d_r ) );
	checkCudaErrors( hipFree( d_p ) );
	checkCudaErrors( hipFree( d_q ) );
	checkCudaErrors( hipFree( d_tmp ) );
}
